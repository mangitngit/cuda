#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <ctime>
#include <iostream>
#include <stdio.h>
#include <cstdlib>

using namespace std;

const int dimension_size = 10240;
const int matrix_size = dimension_size * dimension_size;

const int block_size = 16; 
const int block_size_shared = 16;

__global__ void gpu_transpose(int *matrix_in, int *matrix_out)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(x < dimension_size && y < dimension_size)
		matrix_out[x* dimension_size + y] = matrix_in[y* dimension_size + x];
		//matrix_out[y* dimension_size + x] = matrix_in[x* dimension_size + y];
}

__global__ void gpu_transpose_shared(int *matrix_in, int *matrix_out)
{
	__shared__ int tile[block_size_shared+1][block_size_shared];

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < dimension_size && y < dimension_size)
		tile[threadIdx.x][threadIdx.y] = matrix_in[x * dimension_size + y];

	__syncthreads();	
	if (x < dimension_size && y < dimension_size)
		matrix_out[y * dimension_size + x] = tile[threadIdx.x][threadIdx.y];
}

void print_matrix(int *matrix)
{
	for (int i = 0; i < dimension_size; i++) {
		cout << endl;
		for (int j = 0; j < dimension_size; j++) {
			cout << matrix[i*dimension_size + j] << " ";
		}
	}
	cout << endl;
}

void create_matrix(int *matrix)
{
	printf("Transponowanie macierzy: %d x %d\n\n", dimension_size, dimension_size);

	for (int i = 0; i < dimension_size; i++) {
		for (int j = 0; j < dimension_size; j++) {
			matrix[i*dimension_size + j] = rand() % 50;
		}
	}
}

double cpu_transpose(clock_t start, clock_t end, int *matrix_in, int *matrix_out)
{
	start = clock();
	for (int i = 0; i < dimension_size; i++) {
		for (int j = 0; j < dimension_size; j++) {
			matrix_out[j*dimension_size + i] = matrix_in[i*dimension_size + j];
		}
	}
	end = clock();

	double difference_cpu = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Czas wykonywania CPU: %.4f s\n\n", difference_cpu);

	return difference_cpu;
}

void gpu_transpose(clock_t start, clock_t end, int *matrix_out_gpu, int *dev_matrix_out, int *dev_matrix_in, double difference_cpu)
{
	// USTALANIE WIELKO�CI BLOKU
	int threads_size;
	if (dimension_size % block_size == 0)
		threads_size = (int)(dimension_size / block_size);
	else
		threads_size = (int)(dimension_size / block_size) + 1;

	dim3 blocks(block_size, block_size, 1);
	dim3 threads(threads_size, threads_size, 1);

	start = clock();
	gpu_transpose << <threads, blocks >> > (dev_matrix_in, dev_matrix_out);
	hipDeviceSynchronize();
	hipMemcpy(matrix_out_gpu, dev_matrix_out, matrix_size*sizeof(int), hipMemcpyDeviceToHost);
	end = clock();

	double difference_gpu = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Czas wykonywania GPU: %.4f s\n", difference_gpu);

	hipGetLastError();

	hipMemcpy(dev_matrix_out, matrix_out_gpu, matrix_size*sizeof(int), hipMemcpyHostToDevice);

	printf("Przyspieszenie CPU-GPU:\t\t %.4f - krotne\n\n", difference_cpu / difference_gpu);
}

void gpu_transpose_shared(clock_t start, clock_t end, int *matrix_out_gpu_shared, int *dev_matrix_out, int *dev_matrix_in, double difference_cpu)
{
	// USTALANIE WIELKO�CI BLOKU
	int threads_size;
	if (dimension_size % block_size_shared == 0)
		threads_size = (int)(dimension_size / block_size_shared);
	else
		threads_size = (int)(dimension_size / block_size_shared) + 1;

	dim3 blocks(block_size_shared, block_size_shared, 1);
	dim3 threads(threads_size, threads_size, 1);

	hipMemcpy(dev_matrix_out, matrix_out_gpu_shared, matrix_size*sizeof(int), hipMemcpyHostToDevice);

	start = clock();
	gpu_transpose_shared << <threads, blocks >> > (dev_matrix_in, dev_matrix_out);
	hipDeviceSynchronize();
	hipMemcpy(matrix_out_gpu_shared, dev_matrix_out, matrix_size*sizeof(int), hipMemcpyDeviceToHost);
	end = clock();


	double difference_gpu_shared = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Czas wykonywania GPU: %.4f s\n", difference_gpu_shared);

	hipGetLastError();

	printf("Przyspieszenie CPU-GPU shared:\t %.4f - krotne\n\n", difference_cpu / difference_gpu_shared);
}

void check_matrix(int *matrix_out_cpu, int *matrix_out_gpu, int *matrix_out_gpu_shared)
{
	bool flag = true;
	bool flag_shared = true;
	for (int i = 0; i < dimension_size; i++) {
		for (int j = 0; j < dimension_size; j++) {
			if (matrix_out_cpu[i*dimension_size + j] != matrix_out_gpu[i*dimension_size + j])
				flag = false;
			if (matrix_out_cpu[i*dimension_size + j] != matrix_out_gpu_shared[i*dimension_size + j])
				flag_shared = false;
		}
	}

	cout << "Macierze: " << endl;
	if (flag)
		cout << "CPU-GPU\t\t - takie same" << endl;
	else
		cout << "CPU-GPU\t\t - inne" << endl;

	if (flag_shared)
		cout << "CPU-GPU shared\t - takie same" << endl;
	else
		cout << "CPU-GPU shared\t - inne" << endl;
}

int main()
{
	bool show = false;
	// ZMIENNE
	int *matrix_in;
	int *matrix_out_cpu;
	int *matrix_out_gpu;
	int *matrix_out_gpu_shared;

	matrix_in = (int*)malloc(matrix_size*sizeof(int));
	matrix_out_cpu = (int*)malloc(matrix_size*sizeof(int));
	matrix_out_gpu = (int*)malloc(matrix_size*sizeof(int));
	matrix_out_gpu_shared = (int*)malloc(matrix_size*sizeof(int));

	int *dev_matrix_in;
	int *dev_matrix_out;

	clock_t start = clock();
	clock_t end = clock();

	// TWORZENIE MACIERZY
	create_matrix(matrix_in);
	//show(matrix_in);

	// TRANSPONOWANIE CPU
	double difference_cpu = cpu_transpose(start, end, matrix_in, matrix_out_cpu);
	if(show) print_matrix(matrix_out_cpu);

	// ALOKOWANIE ZMIENNYCH DO GPU
	hipSetDevice(0);

	hipMalloc((void**)&dev_matrix_in, matrix_size*sizeof(int));
	hipMalloc((void**)&dev_matrix_out, matrix_size*sizeof(int));

	hipMemcpy(dev_matrix_in, matrix_in, matrix_size*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_matrix_out, matrix_out_gpu, matrix_size*sizeof(int), hipMemcpyHostToDevice);

	// TRANSPONOWANIE GPU
	gpu_transpose(start, end, matrix_out_gpu, dev_matrix_out, dev_matrix_in, difference_cpu);
	if (show) print_matrix(matrix_out_gpu);
	
	// TRANSPONOWANIE GPU SHARED
	gpu_transpose_shared(start, end, matrix_out_gpu_shared, dev_matrix_out, dev_matrix_in, difference_cpu);
	if (show) print_matrix(matrix_out_gpu_shared);
	
	hipFree(dev_matrix_in);
	hipFree(dev_matrix_out);	
	hipDeviceReset();

	// SPRAWDZENIE POPRAWNIO�CI
	check_matrix(matrix_out_cpu, matrix_out_gpu, matrix_out_gpu_shared);

	free(matrix_in);
	free(matrix_out_cpu);
	free(matrix_out_gpu);
	free(matrix_out_gpu_shared);

	system("pause");
    return 0;
}