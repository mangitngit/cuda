#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>

#include <cstdlib>
#include <assert.h>
#include <stdio.h>
#include <iostream>
#include <ctime>
#include <cmath>
#include <math.h> 

# include "hip/hip_runtime.h"
# include ""
# include <stdio.h>
# include <fstream>
# include <string>
# include <iostream>
# include <stdlib.h>
# include <vector>
# include <ctime>
# include <array>

using namespace std;
int en_de;
string input;
string obraz_cpu;
string obraz_gpu;

void przypisz(int tabin[64], int tabout[64])
{
	for (int i = 0; i < 64; i++) tabout[i] = tabin[i];
}
void podzial(int datatab[64], int tabl[64], int tabr[64])
{
	for (int i = 0; i < 32; i++)
	{
		tabl[i] = datatab[i];
		tabr[i] = datatab[i + 32];

	}
}
void polaczenie(int datatab[64], int tabl[64], int tabr[64])
{
	for (int i = 0; i < 32; i++)
	{
		datatab[i] = tabl[i];
		datatab[i + 32] = tabr[i];

	}
}
void zamiana(int l, int tab[64], int index)
{
	int suma = l;
	for (int i = 3; i >= 0; i--)
	{
		if (suma >= pow(2, i)) {
			suma = suma - pow(2, i);
			tab[3 + index - i] = 1;
		}
		else tab[3 + index - i] = 0;
	}
}

void rysuj(int tabin[64], int ile, int rzadek)
{
	for (int j = 0; j < ile; j++)
	{
		cout << tabin[j] << " ";
		if ((j + 1) % rzadek == 0)cout << endl;
	}
}
void wypelnij(int tabin[64])
{
	for (unsigned long int i = 0; i < 64; i++)
	{
		tabin[i] = i + 1;
	}
}

void IP(int tabin[64], int tabout[64])
{

	for (int i = 0; i < 8; i++)
	{
		tabout[i] = tabin[57 - 8 * i];
		tabout[i + 8] = tabin[59 - 8 * i];
		tabout[i + 16] = tabin[61 - 8 * i];
		tabout[i + 24] = tabin[63 - 8 * i];
		tabout[i + 32] = tabin[56 - 8 * i];
		tabout[i + 40] = tabin[58 - 8 * i];
		tabout[i + 48] = tabin[60 - 8 * i];
		tabout[i + 56] = tabin[62 - 8 * i];
	}
}

void IP_1(int tabin[64], int tabout[64])
{
	for (int i = 0; i < 8; i++)
	{
		tabout[8 * i] = tabin[39 - i];
		tabout[8 * i + 1] = tabin[7 - i];
		tabout[8 * i + 2] = tabin[47 - i];
		tabout[8 * i + 3] = tabin[15 - i];
		tabout[8 * i + 4] = tabin[55 - i];
		tabout[8 * i + 5] = tabin[23 - i];
		tabout[8 * i + 6] = tabin[63 - i];
		tabout[8 * i + 7] = tabin[31 - i];
	}
}

void P_roz(int tabin[64], int tabout[64])
{
	tabout[0] = tabin[31];
	for (int i = 0; i < 5; i++)
	{
		tabout[1 + i] = tabin[i];
		tabout[42 + i] = tabin[27 + i];
	}
	for (int i = 0; i < 6; i++)
	{
		tabout[6 + i] = tabin[i + 3];
		tabout[12 + i] = tabin[7 + i];
		tabout[18 + i] = tabin[11 + i];
		tabout[24 + i] = tabin[15 + i];
		tabout[30 + i] = tabin[19 + i];
		tabout[36 + i] = tabin[23 + i];
	}
	tabout[47] = tabin[0];
}

void xor(int tabr[64], int tabkey[64], int numberofbits)
{
	for (int i = 0; i < numberofbits; i++)
	{
		if (tabr[i] == tabkey[i])tabr[i] = 0;
		else tabr[i] = 1;
	}
}

void PC_1(int tabin[64], int  tabout[64])
{
	for (int i = 0; i < 8; i++)
	{
		tabout[i] = tabin[56 - i * 8];
		tabout[i + 8] = tabin[57 - i * 8];
		tabout[i + 16] = tabin[58 - i * 8];
		tabout[i + 28] = tabin[62 - i * 8];
		tabout[i + 36] = tabin[61 - i * 8];
		tabout[i + 44] = tabin[60 - i * 8];
	}
	for (int i = 0; i < 4; i++)
	{
		tabout[i + 24] = tabin[59 - i * 8];
		tabout[i + 52] = tabin[27 - i * 8];
	}
}

void PC_2(int tabin[64], int  tabout[64])
{
	int  index[48] = { 14, 17, 11, 24, 1, 5, 3, 28, 15, 6, 21, 10, 23, 19, 12, 4, 26, 8, 16, 7, 27, 20, 13, 2, 41, 52, 31, 37, 47, 55, 30, 40, 51, 45, 33, 48, 44, 49, 39, 56, 34, 53, 46, 42,
		50, 36, 29, 32 };
	for (int i = 0; i < 48; i++)
	{
		tabout[i] = tabin[index[i] - 1];
	}
}

void P_P_bloku(int tabin[64], int  tabout[64])
{
	int index[32] = { 16, 7, 20, 21, 29, 12, 28, 17, 1, 15, 23, 26, 5, 18, 31, 10, 2, 8, 24, 14, 32, 27, 3, 9, 19, 13, 30, 6, 22, 11, 4, 25 };
	for (int i = 0; i < 32; i++)
	{
		tabout[i] = tabin[index[i] - 1];
	}
}

void S_blok(int tabin[64], int  tabout[64])
{
	int row = 0;
	int col = 0;
	int ls = 0;
	int ile = 0;
	int S[512] = { 14, 4, 13, 1, 2, 15, 11, 8, 3,
		10,
		6,
		12,
		5,
		9,
		0,
		7, // S1
		0, 15, 7, 4, 14, 2, 13, 1, 10, 6, 12, 11, 9, 5, 3, 8, 4, 1, 14, 8, 13, 6, 2, 11, 15, 12, 9, 7, 3, 10, 5, 0, 15, 12, 8, 2, 4, 9, 1, 7, 5, 11, 3, 14, 10, 0, 6, 13, 15, 1, 8, 14, 6, 11, 3,
		4, 9, 7,
		2,
		13,
		12,
		0,
		5,
		10, // S2
		3, 13, 4, 7, 15, 2, 8, 14, 12, 0, 1, 10, 6, 9, 11, 5, 0, 14, 7, 11, 10, 4, 13, 1, 5, 8, 12, 6, 9, 3, 2, 15, 13, 8, 10, 1, 3, 15, 4, 2, 11, 6, 7, 12, 0, 5, 14, 9, 10, 0, 9, 14, 6, 3, 15,
		5, 1, 13, 12,
		7,
		11,
		4,
		2,
		8, // S3
		13, 7, 0, 9, 3, 4, 6, 10, 2, 8, 5, 14, 12, 11, 15, 1, 13, 6, 4, 9, 8, 15, 3, 0, 11, 1, 2, 12, 5, 10, 14, 7, 1, 10, 13, 0, 6, 9, 8, 7, 4, 15, 14, 3, 11, 5, 2, 12, 7, 13, 14, 3, 0, 6, 9,
		10, 1, 2, 8, 5,
		11,
		12,
		4,
		15, // S4
		13, 8, 11, 5, 6, 15, 0, 3, 4, 7, 2, 12, 1, 10, 14, 9, 10, 6, 9, 0, 12, 11, 7, 13, 15, 1, 3, 14, 5, 2, 8, 4, 3, 15, 0, 6, 10, 1, 13, 8, 9, 4, 5, 11, 12, 7, 2, 14, 2, 12, 4, 1, 7, 10, 11,
		6, 8, 5, 3, 15, 13,
		0,
		14,
		9, // S5
		14, 11, 2, 12, 4, 7, 13, 1, 5, 0, 15, 10, 3, 9, 8, 6, 4, 2, 1, 11, 10, 13, 7, 8, 15, 9, 12, 5, 6, 3, 0, 14, 11, 8, 12, 7, 1, 14, 2, 13, 6, 15, 0, 9, 10, 4, 5, 3, 12, 1, 10, 15, 9, 2, 6,
		8, 0, 13, 3, 4, 14, 7,
		5,
		11, // S6
		10, 15, 4, 2, 7, 12, 9, 5, 6, 1, 13, 14, 0, 11, 3, 8, 9, 14, 15, 5, 2, 8, 12, 3, 7, 0, 4, 10, 1, 13, 11, 6, 4, 3, 2, 12, 9, 5, 15, 10, 11, 14, 1, 7, 6, 0, 8, 13, 4, 11, 2, 14, 15, 0, 8,
		13, 3, 12, 9, 7, 5, 10, 6,
		1, // S7
		13, 0, 11, 7, 4, 9, 1, 10, 14, 3, 5, 12, 2, 15, 8, 6, 1, 4, 11, 13, 12, 3, 7, 14, 10, 15, 6, 8, 0, 5, 9, 2, 6, 11, 13, 8, 1, 4, 10, 7, 9, 5, 0, 15, 14, 2, 3, 12, 13, 2, 8, 4, 6, 15, 11,
		1, 10, 9, 3, 14, 5, 0, 12, 7, // S8
		1, 15, 13, 8, 10, 3, 7, 4, 12, 5, 6, 11, 0, 14, 9, 2, 7, 11, 4, 1, 9, 12, 14, 2, 0, 6, 10, 13, 15, 3, 5, 8, 2, 1, 14, 7, 4, 10, 8, 13, 15, 12, 9, 0, 3, 5, 6, 11 };
	for (int i = 0; i < 8; i++)
	{
		row = tabin[i * 6] * 2 + tabin[i * 6 + 5];
		col = tabin[i * 6 + 1] * 8 + tabin[i * 6 + 2] * 4 + tabin[i * 6 + 3] * 2 + tabin[i * 6 + 4];
		ls = S[row * 16 + col + i * 64];
		ile = i * 4;
		zamiana(ls, tabout, ile);
	}
}

void key_f(int tabkey[64], int ktora, int tabout[64])
{
	int temp = 0;
	int temp1 = 0;
	int temp2 = 0;
	int temp3 = 0;
	int przesuniecie[16] = { 1, 1, 2, 2, 2, 2, 2, 2, 1, 2, 2, 2, 2, 2, 2, 1 };
	if (przesuniecie[ktora] == 1)
	{
		for (int i = 0; i < 28; i++)
		{
			if (i == 0) temp = tabkey[0];
			if (i == 27)tabout[27] = temp;
			else tabout[i] = tabkey[i + 1];
			if (i == 0) temp1 = tabkey[28];
			if (i == 27)tabout[55] = temp1;
			else tabout[i + 28] = tabkey[i + 29];
		}
	}
	if (przesuniecie[ktora] == 2)
	{
		for (int i = 0; i < 28; i++)
		{
			if (i == 0) temp = tabkey[0];
			if (i == 0) temp2 = tabkey[1];
			if (i == 26)tabout[26] = temp;
			else
			{
				if (i == 27)tabout[27] = temp2;
				else tabout[i] = tabkey[i + 2];
			}
			if (i == 0) temp1 = tabkey[28];
			if (i == 0) temp3 = tabkey[29];
			if (i == 26)tabout[54] = temp1;
			else
			{
				if (i == 27)tabout[55] = temp3;
				else tabout[i + 28] = tabkey[i + 30];
			}
		}
	}
}

void funkcja_f(int tabr[64], int tabkey[64], int tabout[64])
{
	P_roz(tabr, tabout);
	xor (tabout, tabkey, 48);
	S_blok(tabout, tabr);
	P_P_bloku(tabr, tabout);
	przypisz(tabout, tabr);
}

void keys(int tabin[64], int tabout[16][64], int temp[64])
{
	PC_1(tabin, temp);
	for (int i = 0; i < 16; i++)
	{
		key_f(temp, i, tabin);
		przypisz(tabin, temp);
		PC_2(tabin, tabout[i]);
	}
}
__device__ void gprzypisz(int tabin[64], int tabout[64])
{
	for (int i = 0; i < 64; i++) tabout[i] = tabin[i];
}
__device__ void gpodzial(int datatab[64], int tabl[64], int tabr[64])
{
	for (int i = 0; i < 32; i++)
	{
		tabl[i] = datatab[i];
		tabr[i] = datatab[i + 32];

	}
}
__device__ void gpolaczenie(int datatab[64], int tabl[64], int tabr[64])
{
	for (int i = 0; i < 32; i++)
	{
		datatab[i] = tabl[i];
		datatab[i + 32] = tabr[i];

	}
}
__device__ void gzamiana(int l, int tab[64], int index)
{
	int potega = 8;
	int suma = l;
	for (int i = 3; i >= 0; i--)
	{
		if (suma >= potega) {
			suma = suma - potega;
			tab[3 + index - i] = 1;
		}
		else tab[3 + index - i] = 0;
		potega = potega / 2;
	}
}

__device__ void gwypelnij(int tabin[64])
{
	for (unsigned long int i = 0; i < 64; i++)
	{
		tabin[i] = i + 1;
	}
}

__device__ void gIP(int tabin[64], int tabout[64])
{

	for (int i = 0; i < 8; i++)
	{
		tabout[i] = tabin[57 - 8 * i];
		tabout[i + 8] = tabin[59 - 8 * i];
		tabout[i + 16] = tabin[61 - 8 * i];
		tabout[i + 24] = tabin[63 - 8 * i];
		tabout[i + 32] = tabin[56 - 8 * i];
		tabout[i + 40] = tabin[58 - 8 * i];
		tabout[i + 48] = tabin[60 - 8 * i];
		tabout[i + 56] = tabin[62 - 8 * i];
	}
}

__device__ void gIP_1(int tabin[64], int tabout[64])
{
	for (int i = 0; i < 8; i++)
	{
		tabout[8 * i] = tabin[39 - i];
		tabout[8 * i + 1] = tabin[7 - i];
		tabout[8 * i + 2] = tabin[47 - i];
		tabout[8 * i + 3] = tabin[15 - i];
		tabout[8 * i + 4] = tabin[55 - i];
		tabout[8 * i + 5] = tabin[23 - i];
		tabout[8 * i + 6] = tabin[63 - i];
		tabout[8 * i + 7] = tabin[31 - i];
	}
}

__device__ void gP_roz(int tabin[64], int tabout[64])
{
	tabout[0] = tabin[31];
	for (int i = 0; i < 5; i++)
	{
		tabout[1 + i] = tabin[i];
		tabout[42 + i] = tabin[27 + i];
	}
	for (int i = 0; i < 6; i++)
	{
		tabout[6 + i] = tabin[i + 3];
		tabout[12 + i] = tabin[7 + i];
		tabout[18 + i] = tabin[11 + i];
		tabout[24 + i] = tabin[15 + i];
		tabout[30 + i] = tabin[19 + i];
		tabout[36 + i] = tabin[23 + i];
	}
	tabout[47] = tabin[0];
}

__device__ void gxor(int tabr[64], int tabkey[64], int numberofbits)
{
	for (int i = 0; i < numberofbits; i++)
	{
		if (tabr[i] == tabkey[i])tabr[i] = 0;
		else tabr[i] = 1;
	}
}

__device__ void gPC_1(int tabin[64], int  tabout[64])
{
	for (int i = 0; i < 8; i++)
	{
		tabout[i] = tabin[56 - i * 8];
		tabout[i + 8] = tabin[57 - i * 8];
		tabout[i + 16] = tabin[58 - i * 8];
		tabout[i + 28] = tabin[62 - i * 8];
		tabout[i + 36] = tabin[61 - i * 8];
		tabout[i + 44] = tabin[60 - i * 8];
	}
	for (int i = 0; i < 4; i++)
	{
		tabout[i + 24] = tabin[59 - i * 8];
		tabout[i + 52] = tabin[27 - i * 8];
	}
}

__device__ void gPC_2(int tabin[64], int  tabout[64])
{
	int  index[48] = { 14, 17, 11, 24, 1, 5, 3, 28, 15, 6, 21, 10, 23, 19, 12, 4, 26, 8, 16, 7, 27, 20, 13, 2, 41, 52, 31, 37, 47, 55, 30, 40, 51, 45, 33, 48, 44, 49, 39, 56, 34, 53, 46, 42,
		50, 36, 29, 32 };
	for (int i = 0; i < 48; i++)
	{
		tabout[i] = tabin[index[i] - 1];
	}
}

__device__ void gP_P_bloku(int tabin[64], int  tabout[64])
{
	int index[32] = { 16, 7, 20, 21, 29, 12, 28, 17, 1, 15, 23, 26, 5, 18, 31, 10, 2, 8, 24, 14, 32, 27, 3, 9, 19, 13, 30, 6, 22, 11, 4, 25 };
	for (int i = 0; i < 32; i++)
	{
		tabout[i] = tabin[index[i] - 1];
	}
}

__device__ void gS_blok(int tabin[64], int  tabout[64])
{
	int row = 0;
	int col = 0;
	int ls = 0;
	int ile = 0;
	int S[512] = { 14, 4, 13, 1, 2, 15, 11, 8, 3,
		10,
		6,
		12,
		5,
		9,
		0,
		7, // S1
		0, 15, 7, 4, 14, 2, 13, 1, 10, 6, 12, 11, 9, 5, 3, 8, 4, 1, 14, 8, 13, 6, 2, 11, 15, 12, 9, 7, 3, 10, 5, 0, 15, 12, 8, 2, 4, 9, 1, 7, 5, 11, 3, 14, 10, 0, 6, 13, 15, 1, 8, 14, 6, 11, 3,
		4, 9, 7,
		2,
		13,
		12,
		0,
		5,
		10, // S2
		3, 13, 4, 7, 15, 2, 8, 14, 12, 0, 1, 10, 6, 9, 11, 5, 0, 14, 7, 11, 10, 4, 13, 1, 5, 8, 12, 6, 9, 3, 2, 15, 13, 8, 10, 1, 3, 15, 4, 2, 11, 6, 7, 12, 0, 5, 14, 9, 10, 0, 9, 14, 6, 3, 15,
		5, 1, 13, 12,
		7,
		11,
		4,
		2,
		8, // S3
		13, 7, 0, 9, 3, 4, 6, 10, 2, 8, 5, 14, 12, 11, 15, 1, 13, 6, 4, 9, 8, 15, 3, 0, 11, 1, 2, 12, 5, 10, 14, 7, 1, 10, 13, 0, 6, 9, 8, 7, 4, 15, 14, 3, 11, 5, 2, 12, 7, 13, 14, 3, 0, 6, 9,
		10, 1, 2, 8, 5,
		11,
		12,
		4,
		15, // S4
		13, 8, 11, 5, 6, 15, 0, 3, 4, 7, 2, 12, 1, 10, 14, 9, 10, 6, 9, 0, 12, 11, 7, 13, 15, 1, 3, 14, 5, 2, 8, 4, 3, 15, 0, 6, 10, 1, 13, 8, 9, 4, 5, 11, 12, 7, 2, 14, 2, 12, 4, 1, 7, 10, 11,
		6, 8, 5, 3, 15, 13,
		0,
		14,
		9, // S5
		14, 11, 2, 12, 4, 7, 13, 1, 5, 0, 15, 10, 3, 9, 8, 6, 4, 2, 1, 11, 10, 13, 7, 8, 15, 9, 12, 5, 6, 3, 0, 14, 11, 8, 12, 7, 1, 14, 2, 13, 6, 15, 0, 9, 10, 4, 5, 3, 12, 1, 10, 15, 9, 2, 6,
		8, 0, 13, 3, 4, 14, 7,
		5,
		11, // S6
		10, 15, 4, 2, 7, 12, 9, 5, 6, 1, 13, 14, 0, 11, 3, 8, 9, 14, 15, 5, 2, 8, 12, 3, 7, 0, 4, 10, 1, 13, 11, 6, 4, 3, 2, 12, 9, 5, 15, 10, 11, 14, 1, 7, 6, 0, 8, 13, 4, 11, 2, 14, 15, 0, 8,
		13, 3, 12, 9, 7, 5, 10, 6,
		1, // S7
		13, 0, 11, 7, 4, 9, 1, 10, 14, 3, 5, 12, 2, 15, 8, 6, 1, 4, 11, 13, 12, 3, 7, 14, 10, 15, 6, 8, 0, 5, 9, 2, 6, 11, 13, 8, 1, 4, 10, 7, 9, 5, 0, 15, 14, 2, 3, 12, 13, 2, 8, 4, 6, 15, 11,
		1, 10, 9, 3, 14, 5, 0, 12, 7, // S8
		1, 15, 13, 8, 10, 3, 7, 4, 12, 5, 6, 11, 0, 14, 9, 2, 7, 11, 4, 1, 9, 12, 14, 2, 0, 6, 10, 13, 15, 3, 5, 8, 2, 1, 14, 7, 4, 10, 8, 13, 15, 12, 9, 0, 3, 5, 6, 11 };
	for (int i = 0; i < 8; i++)
	{
		row = tabin[i * 6] * 2 + tabin[i * 6 + 5];
		col = tabin[i * 6 + 1] * 8 + tabin[i * 6 + 2] * 4 + tabin[i * 6 + 3] * 2 + tabin[i * 6 + 4];
		ls = S[row * 16 + col + i * 64];
		ile = i * 4;
		gzamiana(ls, tabout, ile);
	}
}

__device__ void gkey_f(int tabkey[64], int ktora, int tabout[64])
{
	int temp = 0;
	int temp1 = 0;
	int temp2 = 0;
	int temp3 = 0;
	int przesuniecie[16] = { 1, 1, 2, 2, 2, 2, 2, 2, 1, 2, 2, 2, 2, 2, 2, 1 };
	if (przesuniecie[ktora] == 1)
	{
		for (int i = 0; i < 28; i++)
		{
			if (i == 0) temp = tabkey[0];
			if (i == 27)tabout[27] = temp;
			else tabout[i] = tabkey[i + 1];
			if (i == 0) temp1 = tabkey[28];
			if (i == 27)tabout[55] = temp1;
			else tabout[i + 28] = tabkey[i + 29];
		}
	}
	if (przesuniecie[ktora] == 2)
	{
		for (int i = 0; i < 28; i++)
		{
			if (i == 0) temp = tabkey[0];
			if (i == 0) temp2 = tabkey[1];
			if (i == 26)tabout[26] = temp;
			else
			{
				if (i == 27)tabout[27] = temp2;
				else tabout[i] = tabkey[i + 2];
			}
			if (i == 0) temp1 = tabkey[28];
			if (i == 0) temp3 = tabkey[29];
			if (i == 26)tabout[54] = temp1;
			else
			{
				if (i == 27)tabout[55] = temp3;
				else tabout[i + 28] = tabkey[i + 30];
			}
		}
	}
}

__device__ void gfunkcja_f(int tabr[64], int tabkey[64], int tabout[64])
{
	gP_roz(tabr, tabout);
	gxor(tabout, tabkey, 48);
	gS_blok(tabout, tabr);
	gP_P_bloku(tabr, tabout);
	gprzypisz(tabout, tabr);
}

__device__ void gkeys(int tabin[64], int tabout[16][64], int temp[64])
{
	gPC_1(tabin, temp);
	for (int i = 0; i < 16; i++)
	{
		gkey_f(temp, i, tabin);
		gprzypisz(tabin, temp);
		gPC_2(tabin, tabout[i]);
	}
}
char *dekodowanie(char *Text1, int arraySize)
{

	int total[64];
	int tabkey[64] =
	{
		0,1,0,1,0,1,1,1,
		0,0,1,1,0,1,0,0,
		0,1,0,1,0,1,1,1,
		0,1,1,1,1,0,0,1,
		0,1,0,1,0,1,1,1,
		1,0,1,1,1,1,0,0,
		0,1,0,1,0,1,1,1,
		1,1,1,1,0,0,0,1
	};
	int temp[64];
	int temp1[64];
	int tabr[64];
	int tabl[64];
	int keyss[16][64];

	int i, j, nB, m, iB, k, K, B[8], n, d;
	char *Text = new char[arraySize];
	unsigned char ch;
	Text = Text1;
	i = arraySize;
	keys(tabkey, keyss, temp);
	int mc = 0;

	char *final = new char[arraySize];

	for (iB = 0, nB = 0, m = 0; m<(arraySize / 8); m++) //Repeat for TextLenth/8 times.
	{
		for (iB = 0, i = 0; i<8; i++, nB++)
		{
			ch = Text[nB];
			n = abs((int)ch);//(int)Text[nB];
			for (K = 7; n >= 1; K--)
			{
				B[K] = n % 2;  //Converting 8-Bytes to 64-bit Binary Format
				n /= 2;
			}
			for (; K >= 0; K--) B[K] = 0;
			for (K = 0; K<8; K++, iB++) total[iB] = B[K]; //Now `total' contains the 64-Bit binary format of 8-Bytes
		}

		IP(total, temp);
		podzial(temp, tabl, tabr);//podzial tbr i tbl

		for (i = 15; i > -1; i--)
		{
			przypisz(tabr, temp1);
			funkcja_f(tabr, keyss[i], temp);

			xor (tabr, tabl, 32);
			przypisz(temp1, tabl);

		}
		polaczenie(total, tabr, tabl);
		IP_1(total, temp);

		k = 128;
		d = 0;
		for (i = 0; i<8; i++)
		{
			for (j = 0; j<8; j++)
			{
				d = d + temp[i * 8 + j] * k;
				k = k / 2;
			}
			final[mc++] = (char)d;
			k = 128;
			d = 0;
		}
	} //for loop ends here
	final[mc] = '\0';
	return(final);
}
char *kodowanie(char *Text1, int arraySize)
{

	int total[64];
	int tabkey[64] =
	{
		0,1,0,1,0,1,1,1,
		0,0,1,1,0,1,0,0,
		0,1,0,1,0,1,1,1,
		0,1,1,1,1,0,0,1,
		0,1,0,1,0,1,1,1,
		1,0,1,1,1,1,0,0,
		0,1,0,1,0,1,1,1,
		1,1,1,1,0,0,0,1
	};
	int temp[64];
	int temp1[64];
	int tabr[64];
	int tabl[64];
	int keyss[16][64];
	int i, a1, j, nB, m, iB, k, K, B[8], n, d;
	char *Text = new char[arraySize + 8];
	Text = Text1;
	i = arraySize;
	int mc = 0;
	a1 = i % 8;
	if (a1 != 0) for (j = 0; j<8 - a1; j++, i++) Text[i] = ' ';
	Text[i] = '\0';
	keys(tabkey, keyss, temp);

	arraySize = arraySize + a1;

	char *final = new char[arraySize];

	for (iB = 0, nB = 0, m = 0; m<(arraySize / 8); m++) //Repeat for TextLenth/8 times.
	{
		for (iB = 0, i = 0; i<8; i++, nB++)
		{
			n = (unsigned char)((int)Text[nB]);
			for (K = 7; n >= 1; K--)
			{
				B[K] = n % 2;  //Converting 8-Bytes to 64-bit Binary Format
				n /= 2;

			}
			for (; K >= 0; K--) B[K] = 0;
			for (K = 0; K<8; K++, iB++) total[iB] = B[K]; //Now `total' contains the 64-Bit binary format of 8-Bytes
		}

		IP(total, temp);
		podzial(temp, tabl, tabr);//podzial tbr i tbl

		for (i = 0; i < 16; i++)
		{
			przypisz(tabr, temp1);
			funkcja_f(tabr, keyss[i], temp);

			xor (tabr, tabl, 32);
			przypisz(temp1, tabl);

		}
		polaczenie(total, tabr, tabl);
		IP_1(total, temp);

		k = 128;
		d = 0;

		for (i = 0; i<8; i++)
		{
			for (j = 0; j<8; j++)
			{
				d = d + temp[i * 8 + j] * k;
				k = k / 2;
			}
			final[mc++] = (char)d;
			k = 128;
			d = 0;
		}
	} //for loop ends here

	final[mc] = '\0';
	return(final);
}
__global__ void kodowanie_gpu(char *obraz_in, char *obraz_out, int *dataSize)
{

	int arraySize = 48;
	int total[64];
	int tabkey[64] =
	{
		0,1,0,1,0,1,1,1,
		0,0,1,1,0,1,0,0,
		0,1,0,1,0,1,1,1,
		0,1,1,1,1,0,0,1,
		0,1,0,1,0,1,1,1,
		1,0,1,1,1,1,0,0,
		0,1,0,1,0,1,1,1,
		1,1,1,1,0,0,0,1
	};

	//int thread = blockIdx.x * blockDim.x + threadIdx.x;
	int temp[64];
	int temp1[64];
	int tabr[64];
	int tabl[64];
	int keyss[16][64];
	int  a1, j, nB, m, iB, k, K, B[8], n, d, i;
	//char *Text = new char[arraySize];
	/*
	for (int z = 0; z < arraySize; z++)
	{
	Text[z] = obraz_in[thread*arraySize + z];
	}

	int mc = thread * arraySize;
	keys(tabkey, keyss, temp);*/
	gkeys(tabkey, keyss, temp);
	int thread = ((blockIdx.x * blockDim.x + threadIdx.x));

	if (thread * arraySize > *dataSize)
		return;

	//int ii, a1, jj, nnB, mm, iiB, kk, KK, BB[8], nn, dd, roundround;
	char *Text = new char[arraySize];

	for (int haha = thread * arraySize, int ff = 0; haha < thread * arraySize + arraySize; haha++, ff++)
		Text[ff] = obraz_in[haha];

	i = arraySize;
	int mc = thread * arraySize;
	a1 = i % 8;

	char *final = new char[arraySize];

	for (iB = 0, nB = 0, m = 0; m<(arraySize / 8); m++) //Repeat for TextLenth/8 times.
	{
		for (iB = 0, i = 0; i<8; i++, nB++)
		{
			n = (unsigned char)((int)Text[nB]);
			for (K = 7; n >= 1; K--)
			{
				B[K] = n % 2;  //Converting 8-Bytes to 64-bit Binary Format
				n /= 2;

			}
			for (; K >= 0; K--) B[K] = 0;
			for (K = 0; K<8; K++, iB++) total[iB] = B[K]; //Now `total' contains the 64-Bit binary format of 8-Bytes
		}

		gIP(total, temp);
		gpodzial(temp, tabl, tabr);//podzial tbr i tbl

		for (i = 0; i < 16; i++)
		{
			gprzypisz(tabr, temp1);
			gfunkcja_f(tabr, keyss[i], temp);

			gxor(tabr, tabl, 32);
			gprzypisz(temp1, tabl);

		}
		gpolaczenie(total, tabr, tabl);
		gIP_1(total, temp);

		k = 128;
		d = 0;

		for (i = 0; i<8; i++)
		{
			for (j = 0; j<8; j++)
			{
				d = d + temp[i * 8 + j] * k;
				k = k / 2;
			}
			final[mc++] = (char)d;
			k = 128;
			d = 0;
		}
	} //for loop ends here */
}
char *CUDA1(int dataSize, char *bufor)
{
	int size = dataSize;
	int a1;
	int mc = 0;
	int j;
	a1 = size % 8;
	if (a1 != 0) for (j = 0; j<8 - a1; j++) bufor[j + size] = ' ';
	bufor[j + size] = '\0';
	size = size + a1;
	char *obraz_final;
	obraz_final = (char*)malloc((8 + size) * sizeof(char));

	int rozmiar = 64;
	int blocks = 64;
	int threads = 64;
	char *obraz_in, *obraz_out;
	char *d_obraz_in, *d_obraz_out;
	///////////////////////////////
	int mat_size = rozmiar * sizeof(char);
	int *roz = &size;
	//int *d_roz;
	////////////////////////////////////////////////////
	obraz_in = (char*)malloc(mat_size);
	obraz_out = (char*)malloc(mat_size);
	///////////////////////////////////////////////////////
	hipMalloc(&d_obraz_in, mat_size);
	hipMalloc(&d_obraz_out, mat_size);
	//hipMalloc(&d_roz, sizeof(int));
	for (int i = 0; i <= size / rozmiar; i++)
	{
		for (int j = 0; j < rozmiar && j + i * rozmiar < size; j++) {
			obraz_in[j] = bufor[j + i * rozmiar];
		}


		////////////////////////////////////////////////////////////////////////////////
		hipMemcpy(d_obraz_in, obraz_in, mat_size, hipMemcpyHostToDevice);
		hipMemcpy(d_obraz_out, obraz_in, mat_size, hipMemcpyHostToDevice);
		//hipMemcpy(d_roz, roz, sizeof(int), hipMemcpyHostToDevice);

		//if (en_de) kodowanie_gpu << <blocks, threads>> >(d_obraz_in, d_obraz_out, rozmiar);
		//else dekodowanie_gpu << <blocks, threads >> >(dev_obraz_in, dev_obraz_out, dev_dataSize);

		hipMemcpy(&obraz_out, d_obraz_out, mat_size, hipMemcpyDeviceToHost);
		cout << obraz_out[65];
		for (int k = 0; k < rozmiar; k++);
		obraz_final[j + i * rozmiar] = obraz_out[j];
	}
	obraz_final[size + 7] = '\0';
	return obraz_final;
}
char *CUDA(int dataSize, char *bufor)
{
	char *obraz_final;
	obraz_final = (char*)malloc(dataSize * sizeof(char));

	int rozmiar = 1966080;

	clock_t startGPU = clock();

	for (int ii = 0; ii <= dataSize / rozmiar; ii++)
	{
		char *obraz_in;
		char *obraz_out;
		int *dataS;

		dataS = (int*)malloc(sizeof(int));
		dataS = &dataSize;

		obraz_in = (char*)malloc(dataSize * sizeof(char));
		obraz_out = (char*)malloc(dataSize * sizeof(char));

		char *dev_obraz_in;
		char *dev_obraz_out;
		int *dev_dataSize;

		for (auto i = ii * rozmiar, j = 0; (i <= (rozmiar + ii * rozmiar - 1)) & (i < dataSize); i += 1, j++)
			obraz_in[j] = bufor[i];

		hipSetDevice(0);

		hipMalloc((void**)&dev_obraz_in, dataSize * sizeof(char));
		hipMalloc((void**)&dev_obraz_out, dataSize * sizeof(char));
		hipMalloc((void**)&dev_dataSize, sizeof(int));

		hipMemcpy(dev_obraz_in, obraz_in, dataSize * sizeof(char), hipMemcpyHostToDevice);
		hipMemcpy(dev_obraz_out, obraz_out, dataSize * sizeof(char), hipMemcpyHostToDevice);
		hipMemcpy(dev_dataSize, dataS, sizeof(int), hipMemcpyHostToDevice);

		dim3 threads = 64;
		dim3 blocks = 640;

		kodowanie_gpu << <blocks, threads >> >(dev_obraz_in, dev_obraz_out, dev_dataSize);
		//else decrypt << <blocks, threads >> >(dev_obraz_in, dev_obraz_out, dev_dataSize);

		hipDeviceSynchronize();
		hipMemcpy(obraz_out, dev_obraz_out, dataSize * sizeof(char), hipMemcpyDeviceToHost);

		hipGetLastError();

		hipDeviceReset();

		for (auto i = ii * rozmiar, j = 0; (i <= (rozmiar + ii * rozmiar - 1)) & (i < dataSize); i += 1, j++)
			obraz_final[i] = obraz_out[j];

		hipFree(dev_obraz_in);
		hipFree(dev_obraz_out);
		hipFree(dev_dataSize);
		free(obraz_in);
		free(obraz_out);
	}
	printf("\nCzas wykonywania na GPU: %.4fs\n", (double)(clock() - startGPU) / CLOCKS_PER_SEC);

	return obraz_final;
}

int main()

{
	int arraySize;

	while (true)
	{
		///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

		printf("Kodowanie - 1     Dekodowanie - 0      Wyjscie - 2\nWybor: ");
		cin >> en_de;

		if (en_de == 2) break;

		printf("\nPlik wejsciowy: ");
		cin >> input;
		input.append(".bmp");

		printf("Zapis CPU:      ");
		cin >> obraz_cpu;
		obraz_cpu.append(".bmp");

		printf("Zapis GPU:      ");
		cin >> obraz_gpu;
		obraz_gpu.append(".bmp");

		printf("\nProces w toku:\n");

		////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

		static constexpr size_t HEADER_SIZE = 54;

		ifstream bmp(input, ios::binary);

		ofstream output_cpu(obraz_cpu, ios::binary);

		array<char, HEADER_SIZE> header;
		bmp.read(header.data(), header.size());

		auto fileSize = *reinterpret_cast<uint32_t *>(&header[2]);
		auto dataOffset = *reinterpret_cast<uint32_t *>(&header[10]);
		auto width = *reinterpret_cast<uint32_t *>(&header[18]);
		auto height = *reinterpret_cast<uint32_t *>(&header[22]);
		auto depth = *reinterpret_cast<uint16_t *>(&header[28]);

		vector<char> img(dataOffset - HEADER_SIZE);
		bmp.read(img.data(), img.size());

		int dataSize = ((width * 3 + 3) & (~3)) * height;
		img.resize(dataSize + 1);
		bmp.read(img.data(), img.size());

		char *bufor = new char[dataSize + 1];

		// PRZYPISANIE DO BUFORA
		for (auto i = 0; i <= dataSize - 1; i += 1)
			bufor[i] = img[i];

		// CPU -----------------------
		char *img_output_cpu = new char[dataSize + 1];

		arraySize = dataSize;

		// KODOWANIE I DEKODOWANIE
		clock_t startCPU = clock();

		if (en_de == 1)img_output_cpu = kodowanie(bufor, arraySize);
		if (en_de == 0) img_output_cpu = dekodowanie(bufor, arraySize);

		printf("\nCzas wykonywania na CPU: %.4fs\n", (double)(clock() - startCPU) / CLOCKS_PER_SEC);

		for (int i = 0; i < header.size(); i++)
			output_cpu << header[i];

		for (auto i = 0; i <= dataSize - 1; i += 1)
			output_cpu << img_output_cpu[i];

		output_cpu.close();
		// GPU -----------------------
		char *img_output_gpu = new char[dataSize + 1];
		img_output_gpu = CUDA(arraySize, bufor);

		ofstream output_gpu(obraz_gpu, ios::binary);
		for (int i = 0; i < header.size(); i++)
			output_gpu << header[i];

		for (auto i = 0; i <= dataSize - 1; i += 1)
			output_gpu << img_output_gpu[i];

		output_gpu.close();

		printf("\nNacisnij, aby kontynuowac\n");
		cin.ignore();
		cin.get();

	}



	return 0;
}

