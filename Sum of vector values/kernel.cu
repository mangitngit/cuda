#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <ctime>
#include <iostream>
#include <stdio.h>
#include <cstdlib>

using namespace std;

const unsigned long int vectorsize = 436870912;
const int block_size = 1;
const int thread_size = 1024;

__global__ void reduce_vector(double *vector)
{
	int thread = threadIdx.x;
	__shared__ double bufor[thread_size];

	bufor[thread] = 0;
	for (unsigned long int i = thread; i < vectorsize; i += blockDim.x)
		bufor[thread] += vector[i];

	__syncthreads();

	if (thread == 0)
	{
		for (int i = 1; i < blockDim.x; i++)
			bufor[0] += bufor[i];
		
		vector[0] = bufor[0];
	}
}

void create_vector(double *vector)
{
	for (unsigned long int i = 0; i < vectorsize; i++)
	{
		//double a = (rand() % 44);
		//double b = (rand() % 44)+1;
		//vector[i] = (a / b);
		vector[i] = (rand() % 44);
		//cout << vector[i] << endl;
	}
}

int main()
{
	// ZMIENNE
	double *vector;
	double suma_cpu = 0;
	double suma_gpu = 0;

	vector = (double*)malloc(vectorsize*sizeof(double));

	double *dev_vector;

	clock_t start = clock();
	clock_t end = clock();

	start = clock();
	// WYPE�NIANIE WEKTORA RANDOMAMI
	create_vector(vector);
	end = clock();

		
	printf("Macierz wypelniona\n");

	// SUMOWANIE CPU
	start = clock();
	for (unsigned long int i = 0; i < vectorsize; i++) {
		suma_cpu += vector[i];
	}
	end = clock();


	double difference_cpu = (double)(end - start) / CLOCKS_PER_SEC;
	printf("\nCzas wykonywania CPU: %.4f s\n", difference_cpu);
	printf("Suma = %.2f \n\n", suma_cpu);

	// ALOKOWANIE ZMIENNYCH DO GPU
	hipSetDevice(0);

	hipMalloc((void**)&dev_vector, vectorsize*sizeof(double));
	hipMemcpy(dev_vector, vector, vectorsize*sizeof(double), hipMemcpyHostToDevice);

	start = clock();
	reduce_vector << <block_size, thread_size >> >(dev_vector);
	hipDeviceSynchronize();
	end = clock();
	hipMemcpy(vector, dev_vector, vectorsize*sizeof(double), hipMemcpyDeviceToHost);
	

	hipGetLastError();
	
	suma_gpu = vector[0];

	double difference_gpu = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Czas wykonywania GPU: %.4f s\n", difference_gpu);
	printf("Suma = %.2f \n\n", suma_gpu);

	if (suma_cpu == suma_gpu)
		printf("Wyniki sa takie same\n");
	else
		printf("ERROR ERROR\n");

	printf("Przyspieszenie CPU-GPU:\t\t %.4f - krotne\n\n", difference_cpu / difference_gpu);

	hipFree(dev_vector);

	hipDeviceReset();

	free(vector);
	
	system("pause");
	return 0;
}




