#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cstdlib>
#include <stdio.h>
#include <math.h>
#include <ctime>

#define T 1024 // max threads per block

using namespace std;

uint64_t value_of_number(uint64_t *numer)
{
	uint64_t pierwsza;
	pierwsza = pow(2, 61);

	//uint64_t hehe;
	//hehe = pow(2, 19);
	//hehe = 256203221;
	//hehe = 275604547;
	//hehe = 452930477;
	//hehe = 53;


	*numer = pierwsza - 1;
	//*numer = 53*53;
	//*numer = (hehe - 1)*(hehe - 1);
	//*numer = (hehe)*(hehe);

	//cout << "Podaj liczbe: ";
	//cin >> *numer;

	cout << *numer << endl;
	cout << (uint64_t)sqrt(*numer) << endl << endl;

	return *numer;
}

__global__ void check_prime(uint64_t *numer, bool *flaga)
{
	uint64_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if((*numer % i == 0) && (i >= 2) && (i<*numer))
	{
		printf("Mod %d = %d \n", i, *numer % i);
		*flaga = false;
	}
}

__global__ void check_prime(uint64_t *numer, bool *flaga, int *increase)
{
	uint64_t i = threadIdx.x + blockIdx.x * blockDim.x + *increase*204800000;
	if ((*numer % i == 0) && (i >= 2) && (i<*numer))
	{
		printf("Mod %d = %d \n", i, *numer % i);
		*flaga = false;
	}
}

bool if_prime_cpu(uint64_t x)
{
	if (x < 2)
		return false;
	for (uint64_t i = 2; i*i <= x; i++)
	{
		if (x % i == 0)
		{
			printf("Mod %d = %d \n", i, x % i);
			return false;
		}
	}
	return true;
}

bool if_prime_gpu(bool *flaga, bool *dev_flaga, uint64_t *dev_numer, int *increase, int *dev_increase)
{
	hipMemcpy(dev_increase, increase, sizeof(int), hipMemcpyHostToDevice);

	// Launch a kernel on the GPU
	check_prime << <200000, T >> >(dev_numer, dev_flaga, dev_increase);
	hipDeviceSynchronize();

	hipMemcpy(flaga, dev_flaga, sizeof(bool), hipMemcpyDeviceToHost);

	return *flaga;
}

bool if_prime_gpu(bool *flaga, bool *dev_flaga, uint64_t *dev_numer, uint64_t N)
{
	// Launch a kernel on the GPU
	check_prime << <N, T >> >(dev_numer, dev_flaga);
	hipDeviceSynchronize();

	hipMemcpy(flaga, dev_flaga, sizeof(bool), hipMemcpyDeviceToHost);

	return *flaga;
}

int main()
{
	// zadeklarowanie zmiennych
	uint64_t *numer = new uint64_t;
	uint64_t *sqrt_numer = new uint64_t;

	bool *flaga = new bool;
	int *increase = new int;
	
		*flaga = true;
		*increase = 0;

		// przypisanie warto�ci
		value_of_number(numer);
		*sqrt_numer = (uint64_t)sqrt(*numer);

		// sprawdenie za pomoc� CPU
		clock_t startCPU = clock();
		if (if_prime_cpu(*numer))
			cout << "CPU - tak" << endl;
		else
			cout << "CPU - nie" << endl;
		printf("Czas wykonywania na CPU: %.4fs\n\n", (double)(clock() - startCPU) / CLOCKS_PER_SEC);


		uint64_t N = ceil((uint64_t)sqrt(*numer) / T) + 1;

		uint64_t *dev_numer = 0;
		bool *dev_flaga = false;
		int *dev_increase = 0;

		// Choose which GPU to run on, change this on a multi-GPU system.
		hipSetDevice(0);

		// Allocate GPU buffers for input and output
		hipMalloc((void**)&dev_numer, sizeof(uint64_t));
		hipMalloc((void**)&dev_flaga, sizeof(bool));
		hipMalloc((void**)&dev_increase, sizeof(int));

		// Copy input from host memory to GPU buffers
		hipMemcpy(dev_numer, numer, sizeof(uint64_t), hipMemcpyHostToDevice);
		hipMemcpy(dev_flaga, flaga, sizeof(bool), hipMemcpyHostToDevice);

		// sprawdenie za pomoc� GPU
		clock_t startGPU = clock();
		if (*sqrt_numer > 244800000)
		{
			while (*flaga && (*increase) * 204800000 < *sqrt_numer) {
				*flaga = if_prime_gpu(flaga, dev_flaga, dev_numer, increase, dev_increase);
				*increase = *increase + 1;
			}
		}
		else
		{
			*flaga = if_prime_gpu(flaga, dev_flaga, dev_numer, N);
		}

		if (*flaga)
			cout << "GPU - tak" << endl;
		else
			cout << "GPU - nie" << endl;
		printf("Czas wykonywania na GPU: %.4fs\n", (double)(clock() - startGPU) / CLOCKS_PER_SEC);
	
		hipFree(dev_numer);
		hipFree(dev_flaga);
		hipFree(dev_increase);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipDeviceReset();

	system("pause");
	return 0;
}